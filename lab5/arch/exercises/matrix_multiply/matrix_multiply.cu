#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <chrono>

#define CPU_ITERATIONS 10
#define GPU_ITERATIONS 100

namespace sc = std::chrono;

#define TRY_CUDA() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(-1); \
    }} while(0); \

#define TILE_WIDTH 16

__global__ void gpu_matrix_multiply_simple(float *ma, float *mb, float *mc, size_t width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float result = 0;
	for (int k = 0; k < width; ++k) {
		result += ma[row * width + k] * mb[k * width + col];
	}

	mc[row * width + col] = result;
}

__global__ void gpu_matrix_multiply(float *ma, float *mb, float *mc, size_t width) {
	__shared__ float tile_a[TILE_WIDTH][TILE_WIDTH];
	__shared__ float tile_b[TILE_WIDTH][TILE_WIDTH];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float result = 0;

	for (int k = 0; k < width; k += TILE_WIDTH) {
		tile_a[threadIdx.y][threadIdx.x] = ma[row * width + k + threadIdx.x];
		tile_b[threadIdx.y][threadIdx.x] = mb[(k + threadIdx.y) * width + col];

		__syncthreads();

		for (int i = 0; i < TILE_WIDTH; ++i) {
			result += tile_a[threadIdx.y][i] * tile_b[i][threadIdx.x];
		}

		__syncthreads();
	}

	mc[row * width + col] = result;
}

void cpu_matrix_multiply(float *a, float *b, float *c, size_t width) {
	for (int i = 0; i < width; ++i) {
		for (int j = 0; j < width; ++j) {
			float result = 0;
			for (int k = 0; k < width; ++k) {
				result += a[i * width + k] * b[k * width + j];
			}
			c[i * width + j] = result;
		}
	}
}

void check_results(float *output, float *ref, size_t width) {
	for (int i = 0; i < width; ++i) {
		for (int j = 0; j < width; ++j) {
			if (std::abs(output[i * width + j] - ref[i * width + j]) > 1e-3) {
				std::cerr << "Mismatch at position " << i << ", " << j << ": "
					<< output[i * width + j] << " vs. " << ref[i * width + j] << std::endl;
				return;
			}
		}
	}
}

int main(void) {
	// create a large workload so we can easily measure the
	// performance difference of both implementations

	// note that n measures the width of the matrix, not the number of total
	// elements
	const size_t n = 1 << 10;
	const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
	const dim3 num_blocks(n / block_size.x, n / block_size.y);

	// generate random input on the host
	float *host_a = new float[n * n];
	float *host_b = new float[n * n];
	float *host_c = new float[n * n];
	float *host_out = new float[n * n];

	for (int i = 0; i < n * n; ++i) {
		host_a[i] = static_cast<float>(rand()) / RAND_MAX;
		host_b[i] = static_cast<float>(rand()) / RAND_MAX;
	}

	// allocate storage for the device
	float *device_a = 0, *device_b = 0, *device_c = 0;
	hipMalloc((void **)&device_a, sizeof(float) * n * n);
	hipMalloc((void **)&device_b, sizeof(float) * n * n);
	hipMalloc((void **)&device_c, sizeof(float) * n * n);

	// copy input to the device
	hipMemcpy(device_a, host_a, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(device_b, host_b, sizeof(float) * n * n, hipMemcpyHostToDevice);

	hipEvent_t launch_begin, launch_end;
	hipEventCreate(&launch_begin);
	hipEventCreate(&launch_end);

	// -------- CPU implementation --------

	float average_cpu_time = 0;
	std::cout << "Timing CPU implementation..." << std::endl;

	for (int i = 0; i < CPU_ITERATIONS; ++i) {
		int64_t start = sc::duration_cast<sc::milliseconds>(sc::system_clock::now().time_since_epoch()).count();

		cpu_matrix_multiply(host_a, host_b, host_c, n);

		int64_t end = sc::duration_cast<sc::milliseconds>(sc::system_clock::now().time_since_epoch()).count();

		std::cout << "Iteration " << i << ": " << (end - start) << "ms" << std::endl;
		average_cpu_time += (end - start);
	}

	average_cpu_time /= CPU_ITERATIONS;
	std::cout << "Done." << std::endl;

	// -------- GPU implementation --------

	float average_simple_time = 0;
	std::cout << "Timing simple implementation..." << std::endl;

	for (int i = 0; i < GPU_ITERATIONS; ++i) {
		hipEventRecord(launch_begin);

		gpu_matrix_multiply_simple << <num_blocks, block_size >> > (device_a, device_b, device_c, n);

		hipEventRecord(launch_end);
		hipEventSynchronize(launch_end);

		float time;
		hipEventElapsedTime(&time, launch_begin, launch_end);

		std::cout << "Iteration " << i << ": " << time << "ms" << std::endl;
		average_simple_time += time;

		hipMemcpy(host_out, device_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
		check_results(host_out, host_c, n);
	}

	average_simple_time /= GPU_ITERATIONS;
	std::cout << "Done." << std::endl;

	// -------- GPU optimized implementation --------

	float average_optimized_time = 0;
	std::cout << "Timing optimized implementation..." << std::endl;

	for (int i = 0; i < GPU_ITERATIONS; ++i) {
		hipEventRecord(launch_begin);
		TRY_CUDA();

		gpu_matrix_multiply << <num_blocks, block_size >> > (device_a, device_b, device_c, n);
		TRY_CUDA();

		hipEventRecord(launch_end);
		TRY_CUDA();
		hipEventSynchronize(launch_end);
		TRY_CUDA();

		float time = 0;
		hipEventElapsedTime(&time, launch_begin, launch_end);

		std::cout << "Iteration " << i << ": " << time << "ms" << std::endl;
		average_optimized_time += time;

		hipMemcpy(host_out, device_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
		check_results(host_out, host_c, n);
	}

	average_optimized_time /= GPU_ITERATIONS;
	std::cout << "Done." << std::endl;

	// -------- compute throughput --------

	// report the effective throughput of each kernel in GFLOPS
	// the effective throughput is measured as the number of floating point
	// operations performed per second: (one mul + one add) * N^3
	float cpu_throughput = (2 * n * n * n) / (average_cpu_time / 1000.0f) / 1000000000.0f;
	float simple_throughput = (2 * n * n * n) / (average_simple_time / 1000.0f) / 1000000000.0f;
	float optimized_throughput = (2 * n * n * n) / (average_optimized_time / 1000.0f) / 1000000000.0f;

	std::cout << "Matrix size: " << n << "x" << n << std::endl;
	std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

	std::cout << "Throughput of CPU: " << cpu_throughput << " GFLOPS" << std::endl;
	std::cout << "Throughput of simple kernel: " << simple_throughput << " GFLOPS" << std::endl;
	std::cout << "Throughput of optimized kernel: " << optimized_throughput << " GFLOPS" << std::endl;
	std::cout << std::endl;

	hipEventDestroy(launch_begin);
	hipEventDestroy(launch_end);

	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

	return 0;
}

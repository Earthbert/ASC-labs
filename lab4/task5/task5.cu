#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

__global__ void swap_cuda(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i];
        a[i] = b[i];
        b[i] = c[i];
    }
}

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    // TODO 1: Allocate the host's arrays:
    // host_array_a => 32M
    // host_array_b => 32M
    host_array_a = (int *)malloc(BUF_32M * sizeof(int));
    host_array_b = (int *)malloc(BUF_32M * sizeof(int));

    // TODO 2: Allocate the host's arrays:
    // device_array_a => 32M
    // device_array_b => 32M
    // device_array_c => 2M
    hipMalloc((int **)&device_array_a, BUF_32M * sizeof(int));
    hipMalloc((int **)&device_array_b, BUF_32M * sizeof(int));
    hipMalloc((int **)&device_array_c, BUF_2M * sizeof(int));

    // Check for allocation errors
    if (host_array_a == 0 || host_array_b == 0 ||
        device_array_a == 0 || device_array_b == 0 ||
        device_array_c == 0) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 3: Copy from host to device
    hipMemcpy(device_array_a, host_array_a, BUF_32M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, BUF_32M * sizeof(int), hipMemcpyHostToDevice);

    // TODO 4: Swap the buffers (BUF_2M values each iteration)
    // Hint 1: device_array_c should be used as a temporary buffer
    // Hint 2: hipMemcpy
    int block_size = 256;
    int grid_size = (BUF_2M + block_size - 1) / block_size;
    for (int i = 0; i < BUF_32M; i += BUF_2M) {
        swap_cuda << <grid_size, block_size >> > (device_array_a + i, device_array_b + i, device_array_c, BUF_2M);
    }

    // TODO 5: Copy from device to host
    hipMemcpy(host_array_a, device_array_a, BUF_32M * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_array_b, device_array_b, BUF_32M * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 6: Free the memory
    free(host_array_a);
    free(host_array_b);
    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);

    return 0;
}